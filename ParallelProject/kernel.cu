#include "hip/hip_runtime.h"
#include "Header.h"


__global__ void moveKernel(Point* points, int N, double dT)
{
	//uonic index for each thrad
	//move the point on the fastest way and initlize the clasters
	const int i = blockIdx.x * MAX_NUM_OF_THREADS + threadIdx.x;
	if (i < N)
	{
		points[i].x += dT * points[i].Vx;
		points[i].y += dT * points[i].Vy;
		points[i].z += dT * points[i].Vz;
		points[i].clusterID = 0;
	}
}

__global__ void groupKernel(Cluster* clusters, int K, Point* points, int N, char* flag)
{
	//uonic index for each thrad
	const int Pi = blockIdx.x * MAX_NUM_OF_THREADS + threadIdx.x;
	int Ci;
	double newDistance, oldDistance;

	if (Pi < N)
	{
		for (Ci = 0; Ci < K; Ci++)
		{

			oldDistance = sqrt(pow(points[Pi].x - clusters[points[Pi].clusterID].centerX, 2) + pow(points[Pi].y - clusters[points[Pi].clusterID].centerY, 2) + pow(points[Pi].z - clusters[points[Pi].clusterID].centerZ, 2));
			newDistance = sqrt(pow(points[Pi].x - clusters[Ci].centerX, 2) + pow(points[Pi].y - clusters[Ci].centerY, 2) + pow(points[Pi].z - clusters[Ci].centerZ, 2));
			//calculate distances of points from cluster centers and make switch if necessery
			if (newDistance < oldDistance)
			{
				//if flag ==1 finish the lim for loop
				*flag = 1;
				points[Pi].clusterID = clusters[Ci].ID;
			}
		}

	}
}





// Helper function for using CUDA to add vectors in parallel.
hipError_t cudaRecalculatePoints(Point* points, int N, double dT)
{
	Point *dev_points = 0;
	hipError_t cudaStatus;
	int blocks = N / MAX_NUM_OF_THREADS + 1;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaRecalculatePoints - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(dev_points);
		return cudaStatus;
	}

	// Alloc space for device copies
	cudaStatus = hipMalloc((void**)&dev_points, N * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaRecalculatePoints - hipMalloc failed!");
		hipFree(dev_points);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaRecalculatePoints - hipMemcpy failed!");
		hipFree(dev_points);
		return cudaStatus;
	}


	// Launch a kernel on the GPU with one thread for each element.
	moveKernel << <blocks, MAX_NUM_OF_THREADS >> > (dev_points, N, dT);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "moveKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_points);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaRecalculatePoints - hipDeviceSynchronize returned error code %d after launching moveKernel!\n", cudaStatus);
		hipFree(dev_points);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, N * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaRecalculatePoints - hipMemcpy failed!");
		hipFree(dev_points);
		return cudaStatus;
	}


	hipFree(dev_points);
	return cudaStatus;
}

hipError_t cudaGroupPoints(Cluster* clusters, int K, Point* points, int N, char* flag)
{
	Cluster* dev_clusters = 0;
	Point *dev_points = 0;
	char* dev_flag = 0;
	hipError_t cudaStatus;
	int blocks = N / MAX_NUM_OF_THREADS + 1;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}


	cudaStatus = hipMalloc((void**)&dev_points, N * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMalloc points failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_clusters, K * sizeof(Cluster));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMalloc clusters failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_flag, sizeof(char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMalloc flag failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy input points failed!");
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_clusters, clusters, K * sizeof(Cluster), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy input clusters failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_flag, flag, sizeof(char), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy input flag failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Launch a kernel on the GPU with one thread for each element.
	groupKernel << <blocks, MAX_NUM_OF_THREADS >> > (dev_clusters, K, dev_points, N, dev_flag);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "groupKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching groupKernel!\n", cudaStatus);
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, N * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy output points failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(flag, dev_flag, sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaGroupPoints - hipMemcpy output flag failed!");
		hipFree(dev_flag);
		hipFree(dev_points);
		hipFree(dev_clusters);
		return cudaStatus;
	}

	hipFree(dev_flag);
	hipFree(dev_points);
	hipFree(dev_clusters);
	return cudaStatus;
}

